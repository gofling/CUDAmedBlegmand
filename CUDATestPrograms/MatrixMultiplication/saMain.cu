#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

//#include "kernel.cu"

__global__ void add(int *a, int *b, int *c, int *d, int *e, int *f) {
	*c = *a + *b;
	*d = *a - *b;
	*e = *a * *b;
	*f = *a / *b;
}

int main(){
	
	int a, b, c, d, e, f;
	int *dev_a, *dev_b, *dev_c, *dev_d, *dev_e, *dev_f;
	int size = sizeof(int);

	hipMalloc((void **)&dev_a, size);
	hipMalloc((void **)&dev_b, size);
	hipMalloc((void **)&dev_c, size);
	hipMalloc((void **)&dev_d, size);
	hipMalloc((void **)&dev_e, size);
	hipMalloc((void **)&dev_f, size);

	a = 7;
	b = 2;

	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);

	add << <1, 1 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, dev_f);

	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
	hipMemcpy(&d, dev_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(&e, dev_e, size, hipMemcpyDeviceToHost);
	hipMemcpy(&f, dev_f, size, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);
	hipFree(dev_e);
	hipFree(dev_f);

	printf("Addition: %d\nSubtraction: %d\nMultiplication: %d\nDividation: %d", c, d, e, f);

	return 0;
}