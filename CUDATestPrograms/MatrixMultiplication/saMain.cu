#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <time.h>
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

/* Index to Rows */
#define IDX2R(i,j,ld) (((i)*(ld))+(j))
/* Index to Columns */
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
/* Index to FORTAN (Columns with 1-indexing)*/
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
/* Index to Trinity (Rows with 1-indexing)*/
#define IDX2T(i,j,ld) ((((i)-1)*(ld))+((j)-1))

int main() {
	float h_A[6] = { 1, 2, 3, 4, 5, 6 };
	float h_B[12] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12 };
	int rowsA, colsA, rowsB, colsB;
	rowsA = 2;
	colsA = 3;
	rowsB = 3;
	colsB = 4;
	float h_C[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
	//float *h_C = (float*)calloc(rowsA * colsB, sizeof(float));

	float *d_A, *d_B, *d_C;
	int rowsC = rowsA, colsC = colsB;
	hipError_t error;

	// Allocate memory on Device
	error = hipMalloc(&d_A, rowsA * colsA * sizeof(float));
	if (error != hipSuccess) {
		printf("Memory was not allocated for matrix A");
		return EXIT_FAILURE;
	}

	error = hipMalloc(&d_B, rowsB * colsB * sizeof(float));
	if (error != hipSuccess) {
		printf("Memory was not allocated for matrix B");
		return EXIT_FAILURE;
	}

	error = hipMalloc(&d_C, rowsC * colsC * sizeof(float));
	if (error != hipSuccess) {
		printf("Memory was not allocated for matrix C");
		return EXIT_FAILURE;
	}

	//Copy h_A and h_B to the device
	error = hipMemcpy(d_A, h_A, rowsA * colsA * sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("Copying matrice h_A HtoD failed");
		return EXIT_FAILURE;
	}

	error = hipMemcpy(d_B, h_B, rowsB * colsB * sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("Copying matrice h_B HtoD failed");
		return EXIT_FAILURE;
	}

	// Multiplication on the device
	int lda = rowsA, ldb = colsA, ldc = rowsA;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication (Matrices are stored column-major!)
	if (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsA, colsB, colsA, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc) != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasSgemm failed");
	}

	// Destroy the handle
	hipblasDestroy(handle);

	//Copy result back to the host
	error = hipMemcpy(h_C, d_C, rowsC * colsC * sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("Copying matrix d_C DtoH failed iteration");
		return EXIT_FAILURE;
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();


	for (int i = 0; i < rowsA; ++i){
		for (int j = 0; j < colsB; ++j){
			printf("%f ", h_C[j * rowsA + i]);
		}
		printf("\n");
	}
	printf("\n");

	return 0;
}

//#include "kernel.cu"
/*
__global__ void add(int *a, int *b, int *c, int *d, int *e, int *f) {
	*c = *a + *b;
	*d = *a - *b;
	*e = *a * *b;
	*f = *a / *b;
}

int main(){
	
	int a, b, c, d, e, f;
	int *dev_a, *dev_b, *dev_c, *dev_d, *dev_e, *dev_f;
	int size = sizeof(int);

	hipMalloc((void **)&dev_a, size);
	hipMalloc((void **)&dev_b, size);
	hipMalloc((void **)&dev_c, size);
	hipMalloc((void **)&dev_d, size);
	hipMalloc((void **)&dev_e, size);
	hipMalloc((void **)&dev_f, size);

	a = 7;
	b = 2;

	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);

	add << <1, 1 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, dev_f);

	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
	hipMemcpy(&d, dev_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(&e, dev_e, size, hipMemcpyDeviceToHost);
	hipMemcpy(&f, dev_f, size, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);
	hipFree(dev_e);
	hipFree(dev_f);

	printf("Addition: %d\nSubtraction: %d\nMultiplication: %d\nDividation: %d", c, d, e, f);

	return 0;
}*/