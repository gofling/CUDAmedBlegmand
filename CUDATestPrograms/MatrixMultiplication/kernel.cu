#include <stdio.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>

__global__ void add(int *a, int *b, int *c, int *d, int *e, int *f) {
	*c = *a + *b;
	*d = *a - *b;
	*e = *a * *b;
	*f = *a / *b;
}
