#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <time.h>
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

//Random filling og arrays on device
void GPU_fill_rand(float *A, int nrRowsA, int nrColsA) {

	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	hiprandGenerateUniform(prng, A, nrRowsA * nrColsA);
}

// Function that multiplies matrices on the device 
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda = m, ldb = k, ldc = m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	if (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc) != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasSgemm failed");
	}

	// Destroy the handle
	hipblasDestroy(handle);
}

// Function that multiplies matrices on the host
static void cpu_blas_mmul(int n, const float *A, const float *B, float *C)
{
	const float alpha = 1.0f, beta = 0.0f;
	int i;
	int j;
	int k;

	for (i = 0; i < n; ++i)
	{
		for (j = 0; j < n; ++j)
		{
			float prod = 0;

			for (k = 0; k < n; ++k)
			{
				prod += A[k * n + i] * B[j * n + k];
			}

			C[j * n + i] = alpha * prod + beta * C[j * n + i];
		}
	}
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for (int i = 0; i < nr_rows_A; ++i){
		for (int j = 0; j < nr_cols_A; ++j){
			printf("%f ", A[j * nr_rows_A + i]);
		}

		printf("\n");
	}
	printf("\n");
}

void output_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	FILE *f = fopen("./Output.txt", "w");
	if (f == NULL) {
		printf("Yo dude! Does not load da file!");
		exit(1);
	}
	for (int i = 0; i < nr_rows_A; ++i){
		for (int j = 0; j < nr_cols_A; ++j){
			fprintf(f, "%f ", A[j * nr_rows_A + i]);
		}

		fprintf(f, "\n");
	}
	fprintf(f, "\n\n");

	fclose(f);
}

int main() {

	// 3 Arrays on CPU
	int nrRowsA, nrColsA, nrRowsB, nrColsB, nrRowsC, nrColsC, nrRowsD, nrColsD, sizeAsInput = 1;

	// Square Arrays
	printf("Square matrices of size: ");
	scanf("%d", &sizeAsInput);
	nrRowsA = nrColsA = nrRowsB = nrColsB = nrRowsC = nrColsC = nrRowsD = nrColsD = sizeAsInput;

	// Allocate memory on Host
	clock_t start = clock(), diff;
	float * h_A = (float*)malloc(nrRowsA * nrColsA * sizeof(float));
	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("CPU: A allocation time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	start = clock(), diff;
	float * h_B = (float*)malloc(nrRowsB * nrColsB * sizeof(float));
	diff = clock() - start;
	msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("CPU: B allocation time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	start = clock(), diff;
	float * h_C = (float*)malloc(nrRowsB * nrColsB * sizeof(float));
	diff = clock() - start;
	msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("CPU: C allocation time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	start = clock(), diff;
	float * h_D = (float*)malloc(nrRowsD * nrColsD * sizeof(float));
	diff = clock() - start;
	msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("CPU: D allocation time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	// Allocate memory on Device
	float *d_A, *d_B, *d_C;
	//printf("GPU memory allocation times\n");

	// Memory allocation for Matrix A
	start = clock(), diff;
	if (hipMalloc(&d_A, nrRowsA * nrColsA * sizeof(float)) != hipSuccess) {
		printf("Memory was not allocated for matrix A");
		return EXIT_FAILURE;
	}
	diff = clock() - start;
	msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("A allocation time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	// Memory allocation for Matrix B
	start = clock(), diff;
	if (hipMalloc(&d_B, nrRowsB * nrColsB * sizeof(float)) != hipSuccess) {
		printf("Memory was not allocated for matrix B");
		return EXIT_FAILURE;
	}
	diff = clock() - start;
	msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("B allocation time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	// Memory allocation for Matrix C
	start = clock(), diff;
	if (hipMalloc(&d_C, nrRowsC * nrColsC * sizeof(float)) != hipSuccess) {
		printf("Memory was not allocated for matrix C");
		return EXIT_FAILURE;
	}
	diff = clock() - start;
	msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("C allocation time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);


	// Fill the arrays A and B on GPU with random numbers
	//GPU_fill_rand(d_A, nrRowsA, nrColsA);
	//GPU_fill_rand(d_B, nrRowsB, nrColsB);

	// Optionally we can copy the data back on CPU and print the arrays
	//start = clock(), diff;
	/*if (hipMemcpy(h_A, d_A, nrRowsA * nrColsA * sizeof(float), hipMemcpyDeviceToHost) || hipMemcpy(h_B, d_B, nrRowsB * nrColsB * sizeof(float), hipMemcpyDeviceToHost) != HIPBLAS_STATUS_SUCCESS){
		printf("Copying matrice A or B failed.\n");
		return EXIT_FAILURE;
	}*/
	//diff = clock() - start;
	//msec = diff * 1000 / CLOCKS_PER_SEC;
	//printf("Move random filled arrays from GPU to CPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	//printf("A = \n");
	//print_matrix(h_A, nrRowsA, nrColsA);
	//printf("B = \n");
	//print_matrix(h_B, nrRowsB, nrColsB);

	// Multiply A and B on device
	//start = clock(), diff;
	//gpu_blas_mmul(d_A, d_B, d_C, nrRowsA, nrColsA, nrColsB);
	//diff = clock() - start;
	//msec = diff * 1000 / CLOCKS_PER_SEC;
	//printf("GPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	// Copy (and print) the result on host memory
	//start = clock(), diff;
	//hipMemcpy(h_C, d_C, nrRowsC * nrColsC * sizeof(float), hipMemcpyDeviceToHost);
	//output_matrix(h_C, nrRowsC, nrColsC);
	//diff = clock() - start;
	////msec = diff * 1000 / CLOCKS_PER_SEC;
	//printf("Copy result amtrix from gpu to cpu time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
	//printf("GPU C = \n");
	//print_matrix(h_C, nrRowsC, nrColsC);

	// Multiply A and B on the host
	//start = clock(), diff;
	//cpu_blas_mmul(nrRowsA, h_A, h_B, h_D);
	//diff = clock() - start;
	//msec = diff * 1000 / CLOCKS_PER_SEC;
	//printf("CPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
	//printf("CPU C = \n");
	//print_matrix(h_C, nrRowsC, nrColsC);

	//print_matrix(h_D, nrRowsD, nrColsD);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	//Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	//printf("Done John");
	//int q = 0;
	//scanf("%d", q);

	return 0;
}