#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <time.h>
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

//Random filling og arrays on device
void GPU_fill_rand(float *A, int nrRowsA, int nrColsA) {
	
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	hiprandGenerateUniform(prng, A, nrRowsA * nrColsA);
}

// Function that multiplies matrices on the device 
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda = m, ldb = k, ldc = m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	if (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc) != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasSgemm failed");
	}

	// Destroy the handle
	hipblasDestroy(handle);
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for (int i = 0; i < nr_rows_A; ++i){
		for (int j = 0; j < nr_cols_A; ++j){
			printf("%f ", A[j * nr_rows_A + i]);
		}

		printf("\n");
	}
	printf("\n");
}

void output_matrix(const float *A, int nr_rows_A, int nr_cols_A, char *fileName) {

	FILE *f = fopen(fileName, "a");
	if (f == NULL) {
		printf("Yo dude! Does not load da file!");
		exit(1);
	}
	for (int i = 0; i < nr_rows_A; ++i){
		for (int j = 0; j < nr_cols_A; ++j){
			fprintf(f, "%f, ", A[j * nr_rows_A + i]);
		}

		fprintf(f, "\n");
	}
	fprintf(f, "\n\n");
	
	fclose(f);
}

int main() {
	int nrRowsA, nrColsA, nrRowsB, nrColsB, nrRowsC, nrColsC;
	int matrixStartSize = 500, matrixMaxSize = 100000, sgemmIterations = 2;
	int matrixActualSize = matrixStartSize;
	float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Square Arrays
	nrRowsA = nrColsA = nrRowsB = nrColsB = nrRowsC = nrColsC = matrixActualSize;

	while (matrixActualSize <= matrixMaxSize){
		for (int k = 0; k < sgemmIterations; k++){
			// Allocate memory on Host
			h_A = (float*)malloc(nrRowsA * nrColsA * sizeof(float));
			if (h_A == NULL) { printf("CPU: h_A was not allocated: %d", k); return EXIT_FAILURE; }

			h_B = (float*)malloc(nrRowsB * nrColsB * sizeof(float));
			if (h_B == NULL) { printf("CPU: h_B was not allocated: %d", k); return EXIT_FAILURE; }
		
			h_C = (float*)malloc(nrRowsC * nrColsC * sizeof(float));
			if (h_A == NULL) { printf("CPU: h_C was not allocated: %d", k); return EXIT_FAILURE; }
			
			// Allocate memory on Device

			// Memory allocation for Matrix A
			if (hipMalloc(&d_A, nrRowsA * nrColsA * sizeof(float)) != hipSuccess) {
				printf("Memory was not allocated for matrix A");
				return EXIT_FAILURE;
			}

			// Memory allocation for Matrix B
			if (hipMalloc(&d_B, nrRowsB * nrColsB * sizeof(float)) != hipSuccess) {
				printf("Memory was not allocated for matrix B");
				return EXIT_FAILURE;
			}

			// Memory allocation for Matrix C
			if (hipMalloc(&d_C, nrRowsC * nrColsC * sizeof(float)) != hipSuccess) {
				printf("Memory was not allocated for matrix C");
				return EXIT_FAILURE;
			}

			// Fill the arrays A and B on GPU with random numbers
			GPU_fill_rand(d_A, nrRowsA, nrColsA);
			GPU_fill_rand(d_B, nrRowsB, nrColsB);

			// Optionally we can copy the data back on CPU and print the arrays
			if (hipMemcpy(h_A, d_A, nrRowsA * nrColsA * sizeof(float), hipMemcpyDeviceToHost) || hipMemcpy(h_B, d_B, nrRowsB * nrColsB * sizeof(float), hipMemcpyDeviceToHost) != HIPBLAS_STATUS_SUCCESS){
				printf("Copying matrice A or B failed.\n");
				return EXIT_FAILURE;
			}

			//print_matrix(h_A, nrRowsA, nrColsA);
			//print_matrix(h_B, nrRowsB, nrRowsB);
			//output_matrix(h_A, nrRowsA, nrRowsB, "./matrices.txt");
			//output_matrix(h_B, nrRowsB, nrRowsB, "./matrices1.txt");

			// Multiply A and B on device
			/*clock_t start = clock(), diff;
			gpu_blas_mmul(d_A, d_B, d_C, nrRowsA, nrColsA, nrColsB);
			diff = clock() - start;
			int msec = diff * 1000 / CLOCKS_PER_SEC;
			printf("Warm-up GPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);*/

			clock_t start = clock(), diff;
			gpu_blas_mmul(d_A, d_B, d_C, nrRowsA, nrColsA, nrColsB);
			diff = clock() - start;
			int msec = diff * 1000 / CLOCKS_PER_SEC;
			printf("GPU time, size %d: %d seconds %d milliseconds\n", matrixActualSize, msec / 1000, msec % 1000);

			//start = clock(), diff;
			hipMemcpy(h_C, d_C, nrRowsC * nrColsC * sizeof(float), hipMemcpyDeviceToHost);
			//output_matrix(h_C, nrRowsC, nrColsC, "./matrices.txt");
			//diff = clock() - start;
			////msec = diff * 1000 / CLOCKS_PER_SEC;
			//diff = clock() - start;
			//int msec = diff * 1000 / CLOCKS_PER_SEC;
			//printf("GPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

			// Copy (and print) the result on host memory
			//printf("Copy result amtrix from gpu to cpu time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
			//printf("GPU C = \n");
			//print_matrix(h_C, nrRowsC, nrColsC);

			//diff = clock() - start;
			//msec = diff * 1000 / CLOCKS_PER_SEC;
			//printf("CPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
			//printf("CPU C = \n");
			//print_matrix(h_C, nrRowsC, nrColsC);

			//Free GPU memory
			hipFree(d_A);
			hipFree(d_B);
			hipFree(d_C);

			//Free CPU memory
			free(h_A);
			free(h_B);
			free(h_C);
		}
		matrixActualSize += 500;
	}
	printf("Done John");

	return 0;
}