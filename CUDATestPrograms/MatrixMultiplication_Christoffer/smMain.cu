#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <time.h>
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

//Random filling og arrays on device
void GPU_fill_rand(float *A, int nrRowsA, int nrColsA) {

	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	hiprandGenerateUniform(prng, A, nrRowsA * nrColsA);
}

// Function that multiplies matrices on the device 
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda = m, ldb = k, ldc = m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	if (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc) != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasSgemm failed");
	}

	// Destroy the handle
	hipblasDestroy(handle);
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for (int i = 0; i < nr_rows_A; ++i){
		for (int j = 0; j < nr_cols_A; ++j){
			printf("%f ", A[j * nr_rows_A + i]);
		}

		printf("\n");
	}
	printf("\n");
}

void output_matrix(const float *A, int nr_rows_A, int nr_cols_A, char *fileName) {

	FILE *f = fopen(fileName, "a");
	if (f == NULL) {
		printf("Yo dude! Does not load da file!");
		exit(1);
	}
	for (int i = 0; i < nr_rows_A; ++i){
		for (int j = 0; j < nr_cols_A; ++j){
			fprintf(f, "%ff, ", A[j * nr_rows_A + i]);
		}

		fprintf(f, "\n");
	}
	fprintf(f, "\n\n");
	
	fclose(f);
}

int main() {

	// 3 Arrays on CPU
	int nrRowsA, nrColsA, nrRowsB, nrColsB, nrRowsC, nrColsC;

	// Square Arrays
	nrRowsA = nrColsA = nrRowsB = nrColsB = nrRowsC = nrColsC = 5;

	// Allocate memory on Host
	float * h_A = (float*)malloc(nrRowsA * nrColsA * sizeof(float));
	float * h_B = (float*)malloc(nrRowsB * nrColsB * sizeof(float));
	float * h_C = (float*)malloc(nrRowsB * nrColsB * sizeof(float));

	// Allocate memory on Device
	float *d_A, *d_B, *d_C;
	//printf("GPU memory allocation times\n");

	// Memory allocation for Matrix A
	if (hipMalloc(&d_A, nrRowsA * nrColsA * sizeof(float)) != hipSuccess) {
		printf("Memory was not allocated for matrix A");
		return EXIT_FAILURE;
	}

	// Memory allocation for Matrix B
	if (hipMalloc(&d_B, nrRowsB * nrColsB * sizeof(float)) != hipSuccess) {
		printf("Memory was not allocated for matrix B");
		return EXIT_FAILURE;
	}

	// Memory allocation for Matrix C
	if (hipMalloc(&d_C, nrRowsC * nrColsC * sizeof(float)) != hipSuccess) {
		printf("Memory was not allocated for matrix C");
		return EXIT_FAILURE;
	}

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(d_A, nrRowsA, nrColsA);
	GPU_fill_rand(d_B, nrRowsB, nrColsB);

	// Optionally we can copy the data back on CPU and print the arrays
	if (hipMemcpy(h_A, d_A, nrRowsA * nrColsA * sizeof(float), hipMemcpyDeviceToHost) || hipMemcpy(h_B, d_B, nrRowsB * nrColsB * sizeof(float), hipMemcpyDeviceToHost) != HIPBLAS_STATUS_SUCCESS){
		printf("Copying matrice A or B failed.\n");
		return EXIT_FAILURE;
	}
	
	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(d_A, nrRowsA, nrColsA);
	GPU_fill_rand(d_B, nrRowsB, nrColsB);

	if(hipMemcpy(h_A, d_A, nrRowsA * nrColsA * sizeof(float), hipMemcpyDeviceToHost) || hipMemcpy(h_B, d_B, nrRowsB * nrColsB * sizeof(float), hipMemcpyDeviceToHost) != HIPBLAS_STATUS_SUCCESS){
		printf("Copying matrice A or B failed.\n");
		return EXIT_FAILURE;
	}

	output_matrix(h_A, nrRowsA, nrRowsB, "./matrices.txt");
	output_matrix(h_B, nrRowsB, nrRowsB, "./matrices.txt");

	// Multiply A and B on device
	//start = clock(), diff;
	gpu_blas_mmul(d_A, d_B, d_C, nrRowsA, nrColsA, nrColsB);
	//start = clock(), diff;
	hipMemcpy(h_C, d_C, nrRowsC * nrColsC * sizeof(float), hipMemcpyDeviceToHost);
	output_matrix(h_C, nrRowsC, nrColsC, "./matrices.txt");
	//diff = clock() - start;
	////msec = diff * 1000 / CLOCKS_PER_SEC;
	//diff = clock() - start;
	//msec = diff * 1000 / CLOCKS_PER_SEC;
	//printf("GPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	// Copy (and print) the result on host memory
	//printf("Copy result amtrix from gpu to cpu time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
	//printf("GPU C = \n");
	//print_matrix(h_C, nrRowsC, nrColsC);

	//diff = clock() - start;
	//msec = diff * 1000 / CLOCKS_PER_SEC;
	//printf("CPU time: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
	//printf("CPU C = \n");
	//print_matrix(h_C, nrRowsC, nrColsC);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	//Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	printf("Done John");

	return 0;
}